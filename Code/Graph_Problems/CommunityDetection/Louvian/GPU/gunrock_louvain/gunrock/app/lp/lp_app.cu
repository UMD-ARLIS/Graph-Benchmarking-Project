// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file lp_app.cu
 *
 * @brief Gunrock label propagation (LP) application
 */

#include <gunrock/app/app.cuh>

#include <gunrock/app/lp/lp_problem.cuh>
#include <gunrock/app/lp/lp_enactor.cuh>
#include <gunrock/app/lp/lp_test.cuh>

namespace gunrock {
namespace app {
namespace lp {

hipError_t UseParameters(util::Parameters &parameters) {
  hipError_t retval = hipSuccess;
  GUARD_CU(UseParameters_app(parameters));
  GUARD_CU(UseParameters_problem(parameters));
  GUARD_CU(UseParameters_enactor(parameters));

  GUARD_CU(parameters.Use<std::string>(
      "src",
      util::REQUIRED_ARGUMENT | util::MULTI_VALUE | util::OPTIONAL_PARAMETER,
      "0",
      "<Vertex-ID|random|largestdegree> The source vertices\n"
      "\tIf random, randomly select non-zero degree vertices;\n"
      "\tIf largestdegree, select vertices with largest degrees",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "src-seed",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::OPTIONAL_PARAMETER,
      util::PreDefinedValues<int>::InvalidValue,
      "seed to generate random sources", __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
    "test",
    util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::OPTIONAL_PARAMETER,
    -1,
    "test id for validation", __FILE__, __LINE__));
  
  

  return retval;
}

/**
 * @brief Run LP tests
 * @tparam     GraphT        Type of the graph
 * @tparam     ValueT        Type of the distances
 * @param[in]  parameters    Excution parameters
 * @param[in]  graph         Input graph
 * @param[in]  ref_labels    Reference labels
 * @param[in]  target        Whether to perform the LP
 * \return hipError_t error message(s), if any
 */
template <typename GraphT, typename LabelT = typename GraphT::VertexT>
hipError_t RunTests(util::Parameters &parameters, GraphT &graph,
                     LabelT **ref_labels = NULL,
                     util::Location target = util::DEVICE) {
  hipError_t retval = hipSuccess;
  typedef typename GraphT::VertexT VertexT;
  typedef typename GraphT::SizeT SizeT;
  typedef Problem<GraphT> ProblemT;
  typedef Enactor<ProblemT> EnactorT;
  util::CpuTimer cpu_timer, total_timer;
  cpu_timer.Start();
  total_timer.Start();

  // parse configurations from parameters
  bool quiet_mode = parameters.Get<bool>("quiet");
  int num_runs = parameters.Get<int>("num-runs");
  std::string validation = parameters.Get<std::string>("validation");
  std::vector<VertexT> srcs = parameters.Get<std::vector<VertexT>>("srcs");
  int num_srcs = srcs.size();
  util::Info info("LP", parameters, graph);  // initialize Info structure

  // Allocate host-side array (for both reference and GPU-computed results)
  LabelT *h_labels = new LabelT[graph.nodes];

  // Allocate problem and enactor on GPU, and initialize them
  ProblemT problem(parameters);
  EnactorT enactor;
  GUARD_CU(problem.Init(graph, target));
  GUARD_CU(enactor.Init(problem, target));
  cpu_timer.Stop();
  parameters.Set("preprocess-time", cpu_timer.ElapsedMillis());

  // perform LP
  VertexT src;

  for (int run_num = 0; run_num < num_runs; ++run_num) {
    src = srcs[run_num % num_srcs];
    GUARD_CU(problem.Reset(src, target));
    GUARD_CU(enactor.Reset(src, target));
    util::PrintMsg("__________________________", !quiet_mode);

    cpu_timer.Start();
    GUARD_CU(enactor.Enact(src));
    cpu_timer.Stop();
    info.CollectSingleRun(cpu_timer.ElapsedMillis());

    util::PrintMsg(
        "--------------------------\nRun " + std::to_string(run_num) +
            " elapsed: " + std::to_string(cpu_timer.ElapsedMillis()) +
            " ms, src = " + std::to_string(src) + ", #iterations = " +
            std::to_string(enactor.enactor_slices[0].enactor_stats.iteration),
        !quiet_mode);
    if (validation == "each") {
      GUARD_CU(problem.Extract(h_labels));
      SizeT num_errors = app::lp::Validate_Results(
          parameters, graph, src, h_labels,
          ref_labels == NULL ? NULL : ref_labels[run_num % num_srcs],
          false);
    }
  }

  cpu_timer.Start();
  // Copy out results
  GUARD_CU(problem.Extract(h_labels));
  if (validation == "last") {
    SizeT num_errors = app::lp::Validate_Results(
        parameters, graph, src, h_labels,
        ref_labels == NULL ? NULL : ref_labels[(num_runs - 1) % num_srcs]);
  }

  // compute running statistics
  info.ComputeTraversalStats(enactor, h_labels);
// Display_Memory_Usage(problem);
#ifdef ENABLE_PERFORMANCE_PROFILING
  // Display_Performance_Profiling(&enactor);
#endif

  // Clean up
  GUARD_CU(enactor.Release(target));
  GUARD_CU(problem.Release(target));
  delete[] h_labels;
  h_labels = NULL;
  cpu_timer.Stop();
  total_timer.Stop();

  info.Finalize(cpu_timer.ElapsedMillis(), total_timer.ElapsedMillis());
  return retval;
}

}  // namespace lp
}  // namespace app
}  // namespace gunrock

/*
 * @brief Entry of gunrock_lp function
 * @tparam     GraphT     Type of the graph
 * @tparam     LabelT     Type of the labels
 * @param[in]  parameters Excution parameters
 * @param[in]  graph      Input graph
 * @param[out] labels     Return the labels of the vertices
 * \return     double     Return accumulated elapsed times for all runs
 */
template <typename GraphT, typename LabelT = typename GraphT::VertexT>
double gunrock_lp(gunrock::util::Parameters &parameters, GraphT &graph,
                   LabelT **labels) {
  typedef typename GraphT::VertexT VertexT;
  typedef gunrock::app::lp::Problem<GraphT> ProblemT;
  typedef gunrock::app::lp::Enactor<ProblemT> EnactorT;
  gunrock::util::CpuTimer cpu_timer;
  gunrock::util::Location target = gunrock::util::DEVICE;
  double total_time = 0;
  if (parameters.UseDefault("quiet")) parameters.Set("quiet", true);
  // Allocate problem and enactor on GPU, and initialize them
  ProblemT problem(parameters);
  EnactorT enactor;
  problem.Init(graph, target);
  enactor.Init(problem, target);

  std::vector<VertexT> srcs = parameters.Get<std::vector<VertexT>>("srcs");
  int num_runs = parameters.Get<int>("num-runs");
  int num_srcs = srcs.size();
  for (int run_num = 0; run_num < num_runs; ++run_num) {
    int src_num = run_num % num_srcs;
    VertexT src = srcs[src_num];
    problem.Reset(src, target);
    enactor.Reset(src, target);

    cpu_timer.Start();
    enactor.Enact(src);
    cpu_timer.Stop();

    total_time += cpu_timer.ElapsedMillis();
    problem.Extract(labels[src_num]);
  }

  enactor.Release(target);
  problem.Release(target);
  srcs.clear();
  return total_time;
}

/*
 * @brief Simple interface take in graph as CSR format
 * @param[in]  num_nodes   Number of veritces in the input graph
 * @param[in]  num_edges   Number of edges in the input graph
 * @param[in]  row_offsets CSR-formatted graph input row offsets
 * @param[in]  col_indices CSR-formatted graph input column indices
 * @param[in]  edge_values CSR-formatted graph input edge weights
 * @param[out] labels      Return shortest hop distances to source per vertex
 * \return     double      Return accumulated elapsed times for all runs
 */
template <typename VertexT = int, typename SizeT = int,
          typename LabelT = VertexT>
double lp(const SizeT num_nodes, const SizeT num_edges,
           const SizeT *row_offsets, const VertexT *col_indices,
           const int num_runs, VertexT *sources, LabelT **labels) {
  typedef typename gunrock::app::TestGraph<VertexT, SizeT, VertexT,
                                           gunrock::graph::HAS_CSR |
                                               gunrock::graph::HAS_CSC>
      GraphT;
  typedef typename GraphT::CsrT CsrT;

  // Setup parameters
  gunrock::util::Parameters parameters("lp");
  gunrock::graphio::UseParameters(parameters);
  gunrock::app::lp::UseParameters(parameters);
  gunrock::app::UseParameters_test(parameters);
  parameters.Parse_CommandLine(0, NULL);
  parameters.Set("graph-type", "by-pass");
  parameters.Set("num-runs", num_runs);
  parameters.Set("test", -1);
  std::vector<VertexT> srcs;
  for (int i = 0; i < num_runs; i++) srcs.push_back(sources[i]);
  parameters.Set("srcs", srcs);

  bool quiet = parameters.Get<bool>("quiet");
  GraphT graph;
  // Assign pointers into gunrock graph format
  graph.CsrT::Allocate(num_nodes, num_edges, gunrock::util::HOST);
  graph.CsrT::row_offsets.SetPointer((SizeT *)row_offsets, num_nodes + 1,
                                     gunrock::util::HOST);
  graph.CsrT::column_indices.SetPointer((VertexT *)col_indices, num_edges,
                                        gunrock::util::HOST);
  graph.FromCsr(graph.csr(), gunrock::util::HOST, 0, quiet, true);
  gunrock::graphio::LoadGraph(parameters, graph);

  // Run the LP
  double elapsed_time = gunrock_lp(parameters, graph, labels);

  // Cleanup
  graph.Release();
  srcs.clear();

  return elapsed_time;
}

/*
 * @brief Simple C-interface take in graph as CSR format
 * @param[in]  num_nodes   Number of veritces in the input graph
 * @param[in]  num_edges   Number of edges in the input graph
 * @param[in]  row_offsets CSR-formatted graph input row offsets
 * @param[in]  col_indices CSR-formatted graph input column indices
 * @param[out] labels      Return shortest hop distances to source per vertex
 * \return     double      Return accumulated elapsed times for all runs
 */
double lp(const int num_nodes, const int num_edges, const int *row_offsets,
           const int *col_indices, int source,
           int *distances) {
  return lp(num_nodes, num_edges, row_offsets, col_indices, 1, &source, &distances);
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
