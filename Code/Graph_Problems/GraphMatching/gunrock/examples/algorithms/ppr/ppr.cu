#include "hip/hip_runtime.h"
#include <gunrock/algorithms/ppr.hxx>
#include "ppr_cpu.hxx"

using namespace gunrock;
using namespace memory;

void test_ppr(int num_arguments, char** argument_array) {
  if (num_arguments != 2) {
    std::cerr << "usage: ./bin/<program-name> filename.mtx" << std::endl;
    exit(1);
  }

  // --
  // Define types

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;

  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;
  csr_t csr;

  // --
  // IO

  weight_t alpha = 0.15;
  weight_t epsilon = 1e-6;
  vertex_t n_seeds = 10;

  std::string filename = argument_array[1];

  if (util::is_market(filename)) {
    io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
    csr.from_coo(mm.load(filename));
  } else if (util::is_binary_csr(filename)) {
    csr.read_binary(filename);
  } else {
    std::cerr << "Unknown file format: " << filename << std::endl;
    exit(1);
  }

  // --
  // Build graph

  auto G = graph::build::from_csr<memory_space_t::device, graph::view_t::csr>(
      csr.number_of_rows,               // rows
      csr.number_of_columns,            // columns
      csr.number_of_nonzeros,           // nonzeros
      csr.row_offsets.data().get(),     // row_offsets
      csr.column_indices.data().get(),  // column_indices
      csr.nonzero_values.data().get()   // values
  );  // supports row_indices and column_offsets (default = nullptr)

  // --
  // Params and memory allocation

  vertex_t n_vertices = G.get_number_of_vertices();

  thrust::device_vector<weight_t> p(n_seeds * n_vertices);

  // --
  // GPU Run

  float gpu_elapsed =
      gunrock::ppr::run_batch(G, n_seeds, p.data().get(), alpha, epsilon);

  // --
  // CPU Run

  thrust::host_vector<weight_t> h_p(n_seeds * n_vertices);

  float cpu_elapsed = ppr_cpu::run<csr_t, vertex_t, edge_t, weight_t>(
      csr, n_seeds, h_p.data(), alpha, epsilon);

  int n_errors = util::compare(p.data().get(), h_p.data(), n_seeds * n_vertices,
                               [epsilon](const weight_t a, const weight_t b) {
                                 return std::abs(a - b) > epsilon;
                               });

  // --
  // Log + Validate

  print::head(p, 40, "GPU rank");
  print::head(h_p, 40, "CPU rank");

  std::cout << "GPU Elapsed Time : " << gpu_elapsed << " (ms)" << std::endl;
  std::cout << "CPU Elapsed Time : " << cpu_elapsed << " (ms)" << std::endl;
  std::cout << "Number of errors : " << n_errors << std::endl;
}

int main(int argc, char** argv) {
  test_ppr(argc, argv);
}
