#include "hip/hip_runtime.h"
#include <gunrock/algorithms/spmv.hxx>
#include <gunrock/algorithms/generate/random.hxx>
#include "spmv_cpu.hxx"

using namespace gunrock;
using namespace memory;

void test_spmv(int num_arguments, char** argument_array) {
  if (num_arguments != 2) {
    std::cerr << "usage: ./bin/<program-name> filename.mtx" << std::endl;
    exit(1);
  }

  // --
  // Define types
  // Specify the types that will be used for
  // - vertex ids (vertex_t)
  // - edge offsets (edge_t)
  // - edge weights (weight_t)

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;

  // --
  // IO

  // Filename to be read
  std::string filename = argument_array[1];

  // Load the matrix-market dataset into csr format.
  // See `format` to see other supported formats.
  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;
  csr_t csr;
  csr.from_coo(mm.load(filename));

  // --
  // Build graph

  // Convert the dataset you loaded into an `essentials` graph.
  // `memory_space_t::device` -> the graph will be created on the GPU.
  // `graph::view_t::csr`     -> your input data is in `csr` format.
  //
  // Note that `graph::build::from_csr` expects pointers, but the `csr` data
  // arrays are `thrust` vectors, so we need to unwrap them w/ `.data().get()`.
  auto G = graph::build::from_csr<memory_space_t::device, graph::view_t::csr>(
      csr.number_of_rows, csr.number_of_columns, csr.number_of_nonzeros,
      csr.row_offsets.data().get(), csr.column_indices.data().get(),
      csr.nonzero_values.data().get());

  // --
  // Params and memory allocation
  vertex_t n_vertices = G.get_number_of_vertices();
  thrust::device_vector<weight_t> x(n_vertices);
  thrust::device_vector<weight_t> y(n_vertices);

  gunrock::generate::random::uniform_distribution(x);

  // --
  // GPU Run
  float gpu_elapsed = gunrock::spmv::run(G, x.data().get(), y.data().get());

  // --
  // CPU Run

  thrust::host_vector<weight_t> y_h(n_vertices);
  float cpu_elapsed = spmv_cpu::run(csr, x, y_h);

  // --
  // Log + Validate
  int n_errors = util::compare(
      y.data().get(), y_h.data(), n_vertices,
      [=](const weight_t a, const weight_t b) {
        // TODO: needs better accuracy.
        return std::abs(a - b) > 1e-2;
      },
      true);

  gunrock::print::head(y, 40, "GPU y-vector");
  gunrock::print::head(y_h, 40, "CPU y-vector");

  std::cout << "GPU Elapsed Time : " << gpu_elapsed << " (ms)" << std::endl;
  std::cout << "CPU Elapsed Time : " << cpu_elapsed << " (ms)" << std::endl;
  std::cout << "Number of errors : " << n_errors << std::endl;
}

// Main method, wrapping test function
int main(int argc, char** argv) {
  test_spmv(argc, argv);
}