#include <nvbench/nvbench.cuh>
#include <cxxopts.hpp>
#include <gunrock/algorithms/algorithms.hxx>
#include <gunrock/algorithms/spgemm.hxx>

using namespace gunrock;
using namespace memory;

using vertex_t = int;
using edge_t = int;
using weight_t = float;

std::string filename_a;
std::string filename_b;

struct parameters_t {
  std::string filename_a;
  std::string filename_b;
  bool help = false;
  cxxopts::Options options;

  /**
   * @brief Construct a new parameters object and parse command line arguments.
   *
   * @param argc Number of command line arguments.
   * @param argv Command line arguments.
   */
  parameters_t(int argc, char** argv)
      : options(argv[0], "SPGEMM Benchmarking") {
    options.allow_unrecognised_options();
    // Add command line options
    options.add_options()("h,help", "Print help")  // help
        ("a,amatrix", "Matrix A file",
         cxxopts::value<std::string>())  // mtx A
        ("b,bmatrix", "Matrix B file",
         cxxopts::value<std::string>());  // mtx B

    // Parse command line arguments
    auto result = options.parse(argc, argv);

    if (result.count("help")) {
      help = true;
      std::cout << options.help({""});
      std::cout << "  [optional nvbench args]" << std::endl << std::endl;
      // Do not exit so we also print NVBench help.
    } else {
      if (result.count("amatrix") == 1) {
        filename_a = result["amatrix"].as<std::string>();
        if (!util::is_market(filename_a)) {
          std::cout << options.help({""});
          std::cout << "  [optional nvbench args]" << std::endl << std::endl;
          std::exit(0);
        }
      } else {
        std::cout << options.help({""});
        std::cout << "  [optional nvbench args]" << std::endl << std::endl;
        std::exit(0);
      }
      if (result.count("bmatrix") == 1) {
        filename_b = result["bmatrix"].as<std::string>();
        if (!util::is_market(filename_b)) {
          std::cout << options.help({""});
          std::cout << "  [optional nvbench args]" << std::endl << std::endl;
          std::exit(0);
        }
      } else {
        std::cout << options.help({""});
        std::cout << "  [optional nvbench args]" << std::endl << std::endl;
        std::exit(0);
      }
    }
  }
};

void spgemm_bench(nvbench::state& state) {
  // --
  // Add metrics
  state.collect_dram_throughput();
  state.collect_l1_hit_rates();
  state.collect_l2_hit_rates();
  state.collect_loads_efficiency();
  state.collect_stores_efficiency();

  // --
  // Define types
  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;

  // --
  // Build graphs + metadata
  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
  csr_t a_csr;
  a_csr.from_coo(mm.load(filename_a));

  auto A = graph::build::from_csr<memory_space_t::device, graph::view_t::csr>(
      a_csr.number_of_rows, a_csr.number_of_columns, a_csr.number_of_nonzeros,
      a_csr.row_offsets.data().get(), a_csr.column_indices.data().get(),
      a_csr.nonzero_values.data().get());

  csr_t b_csr;
  b_csr.from_coo(mm.load(filename_b));

  auto B_csr =
      graph::build::from_csr<memory_space_t::device, graph::view_t::csr>(
          b_csr.number_of_rows, b_csr.number_of_columns,
          b_csr.number_of_nonzeros, b_csr.row_offsets.data().get(),
          b_csr.column_indices.data().get(), b_csr.nonzero_values.data().get());

  thrust::device_vector<vertex_t> row_indices(b_csr.number_of_nonzeros);
  thrust::device_vector<edge_t> column_offsets(b_csr.number_of_columns + 1);

  auto B = graph::build::from_csr<memory_space_t::device, graph::view_t::csc>(
      b_csr.number_of_rows, b_csr.number_of_columns, b_csr.number_of_nonzeros,
      b_csr.row_offsets.data().get(), b_csr.column_indices.data().get(),
      b_csr.nonzero_values.data().get(), row_indices.data().get(),
      column_offsets.data().get());

  csr_t C;

  // --
  // Run SPGEMM with NVBench
  state.exec(nvbench::exec_tag::sync, [&](nvbench::launch& launch) {
    gunrock::spgemm::run(A, B_csr, B, C);
  });
}

int main(int argc, char** argv) {
  parameters_t params(argc, argv);
  filename_a = params.filename_a;
  filename_b = params.filename_b;

  if (params.help) {
    // Print NVBench help.
    const char* args[1] = {"-h"};
    NVBENCH_MAIN_BODY(1, args);
  } else {
    // Create a new argument array without matrix filenames to pass to NVBench.
    char* args[argc - 4];
    int j = 0;
    for (int i = 0; i < argc; i++) {
      if (strcmp(argv[i], "--amatrix") == 0 || strcmp(argv[i], "-a") == 0 ||
          strcmp(argv[i], "--bmatrix") == 0 || strcmp(argv[i], "-b") == 0) {
        i++;
        continue;
      }
      args[j] = argv[i];
      j++;
    }

    NVBENCH_BENCH(spgemm_bench);
    NVBENCH_MAIN_BODY(argc - 4, args);
  }
}
